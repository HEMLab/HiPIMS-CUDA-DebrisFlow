#include "hip/hip_runtime.h"
// ======================================================================================
// Name                :    High-Performance Integrated Modelling System
// Description         :    This code pack provides a generic framework for developing 
//                          Geophysical CFD software. Legacy name: GeoClasses
// ======================================================================================
// Version             :    1.0.1 
// Author              :    Xilin Xia
// Create Time         :    2014/10/04
// Update Time         :    2020/04/26
// ======================================================================================
// LICENCE: GPLv3 
// ======================================================================================

/*!
  \file cuda_cal_metric.cu
  \brief Source file for writing output as arcgis ascii files

*/

#include "cuda_cal_metric.h"

namespace GC{
  __global__ void cuCalDepthDurationKernel(Scalar* h_old, Scalar* h, Scalar* t_hGTx, Scalar x, Scalar dt, unsigned int phi_size){

    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    while(index < phi_size){
      Scalar t = t_hGTx[index];
      if(h_old[index] > x &&  h[index] > x){
        t += dt;
      }else{
        t = 0.0;
      }
      t_hGTx[index] = t;
      index += blockDim.x * gridDim.x;
    }

  }

  void cuCalDepthDuration(cuFvMappedField<Scalar, on_cell>& h_old, cuFvMappedField<Scalar, on_cell>& h, cuFvMappedField<Scalar, on_cell>& t_hGTx, Scalar x, Scalar dt){

    cuCalDepthDurationKernel << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(h_old.data.dev_ptr(),
    h.data.dev_ptr(),
    t_hGTx.data.dev_ptr(),
    x,
    dt,
    h.data.size());

  }

  __global__ void cuCalHazardRatingKernel(Scalar* h, Vector* hU, Scalar* hRating, unsigned int phi_size){

    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    while(index < phi_size){
      Scalar h_ = h[index];
      Vector2 hU_ = hU[index];
      Scalar debris_factor = 0.0;
      if(h_> 0.25){
        debris_factor = 1.0;
      }else{
        debris_factor = 0.5;
      }
      Scalar h_rating = norm(hU_) + 0.5*h_ + debris_factor;
      hRating[index] = h_rating;
      index += blockDim.x * gridDim.x;
    }

  }

  void cuCalHazardRating(cuFvMappedField<Scalar, on_cell>& h, cuFvMappedField<Vector, on_cell>& hU, cuFvMappedField<Scalar, on_cell>& hRating){

    cuCalHazardRatingKernel << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(h.data.dev_ptr(),
    hU.data.dev_ptr(),
    hRating.data.dev_ptr(),
    h.data.size());

  }

}